/*
* ARQUITECTURA DE COMPUTADORES
* 2� Grado en Ingenieria Informatica
*
* PRACTICA 2: "Suma De Matrices Paralela"
* >> Arreglar for en __global__
* >> Pasar numElem como argumento
*
* AUTOR: Ivanes
*/
///////////////////////////////////////////////////////////////////////////
// Includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


// Defines
#define RAN_MIN 1
#define RAN_MAX 5

// Declaracion de funciones
void cudaDev()
{
	// Saca num hilos, funcion CUDA
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	//
	printf("\n***********************************************************************\n\n");
	printf("> Nombre Dispositivos: %s\n", deviceProp.name);
	printf("> Capacidad de Computo: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("> Numero de MultiProcesadores: %d \n", deviceProp.multiProcessorCount);
	printf("> Numero de Nucleos (Arq. PASCAL): %d \n", 64);
	printf("> Maximo de hilos por eje en bloque\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("> Maximo de bloques por eje\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("\n***********************************************************************\n");
}

__global__
void mueveFila(int *dev_matriz, int *dev_matriz_resultado, int numFilas)
{
	int columna = threadIdx.x;
	int fila = threadIdx.y;

	// Indice normal
	int myID = fila + columna * blockDim.x;

	// Indice sumandole una fila, y restandole una columna en la ultima fila
	if(fila == (numFilas-1))
		columna--;

	int myID_2 = (fila+1) + columna * blockDim.x;

	dev_matriz_resultado[myID_2] = dev_matriz[myID];
}

// MAIN: Rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	//Eventos
	hipEvent_t start;
	hipEvent_t stop;

	// Creacion de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Marca de inicio
	hipEventRecord(start, 0);

	// Declaracion
	int *hst_matriz;
	int *hst_matriz_resultado;
	int *dev_matriz;
	int *dev_matriz_resultado;

	// Filas y Columnas
	int filas;
	int columnas;

	// Llama a la funci�n Cuda que devuelve info
	cudaDev();

	// Pregunta n�mero de filas y columnas
	printf("Numero maximo de elementos: 1024");
	do {
		printf("\n\nNumero de filas: ");
		scanf("%d", &filas);
		getchar();

		printf("\nNumero de columnas: ");
		scanf("%d", &columnas);
		getchar();
	} while ((filas*columnas > 1024) || filas <= 0 || columnas <= 0);

	// Saca el tama�o del array
	printf("\nNumero de elementos: %d", filas*columnas);

	// Dimensiones del kernel
	dim3 Nbloques(1);
	dim3 hilosB(columnas, filas);

	// Reserva en el host
	hst_matriz = (int*)malloc(filas*columnas * sizeof(int));
	hst_matriz_resultado = (int*)malloc(filas*columnas * sizeof(int));

	// Reserva en el device
	hipMalloc( &dev_matriz, filas*columnas * sizeof(int));
	hipMalloc( &dev_matriz_resultado, filas*columnas * sizeof(int));

	// Insertamos valores random en la matriz
	srand((int)time(NULL));
	for (int i = 0; i < filas*columnas; i++)
	{
		hst_matriz[i] = RAN_MIN + rand() % RAN_MAX;
	}

	// Pasamos el array al device y le damos la vuelta
	hipMemcpy(dev_matriz, hst_matriz, filas*columnas * sizeof(int), hipMemcpyHostToDevice);
	mueveFila <<<Nbloques, hilosB>>>(dev_matriz, dev_matriz_resultado, filas);

	// Check de errores
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	// Pasamos el resultado a la cpu
	hipMemcpy(hst_matriz_resultado, dev_matriz_resultado, filas*columnas * sizeof(int), hipMemcpyDeviceToHost);

	// Muestra contenido de arrays y resultado
	printf("\n\nMatriz: \n");
	printf("*****************\n");

	for (int i = 0; i < filas; i++) {
		for(int j = 0; j < columnas; j++) {
			printf("%d ", hst_matriz[i+j*columnas]);
		}
		printf("\n");
	}

	printf("\n\nMatriz Resultado: \n");
	printf("*********************\n");

	for (int i = 0; i < filas; i++) {
		for(int j = 0; j < columnas; j++) {
			printf("%d ", hst_matriz_resultado[i+j*columnas]);
		}
		printf("\n");
	}
	
	// Marca de final
	hipEventRecord(stop, 0);

	// Sincronizacion CPU-GPU
	hipEventSynchronize(stop);

	// Calculo del tiempo
	float tiempoTrans;
	hipEventElapsedTime(&tiempoTrans, start, stop);
	printf("\n\n> Tiempo de ejecuccion: %f ms\n", tiempoTrans);

	// Liberacion de recursos
	free(hst_matriz);
	free(hst_matriz_resultado);
	hipFree(dev_matriz);
	hipFree(dev_matriz_resultado);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Salida
	time_t fecha;
	time(&fecha);
	printf("\n\n***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
